#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define N 1024
#define BLOCK_SIZE 16 //most popular, esp for tiling

void cpu_matrixmul(int n, float* a, float* b, float* c) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            float sum = 0.0f;
            for (int k = 0; k < n; k++) {
                sum += a[i * n + k] * b[k * n + j];
                // sum += A[i][k] * B[k][j]
            }
            c[i * n + j] = sum;
        }
    }
}

__global__ 
void cuda_matrixmul(int n, float* a, float*b, float* c) {
    int col = blockIdx.x * blockDim.x + threadIdx.x; //x goes across columns
    int row = blockIdx.y * blockDim.y + threadIdx.y; //y goes across rows
    //row and column correspond to c[row][col] so does entire computation for this

    if (row < n && col < n) { //this runs sequentially
        float sum = 0.0f;
        for (int k = 0; k < n; k++) {
            sum += a[row * n + k] * b[k * n + col];
                // sum += A[i][k] * B[k][j]
        }
        c[row * n + col] = sum;
    }
}

__global__
void cuda_tiled_matrixmul(int n, float* a, float* b, float* c) {
    __shared__ float tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * BLOCK_SIZE + ty; //global row for c
    int col = blockIdx.x * BLOCK_SIZE + tx; //global col for c

    float sum = 0.0f;

    for (int tile = 0; tile < ((n + BLOCK_SIZE - 1) / BLOCK_SIZE); tile++) {
        //load tile_a
        int a_col = tile * BLOCK_SIZE + tx;
        if (row < n && a_col < n) {
            tile_a[ty][tx] = a[row * n + a_col];
        }
        else {
            tile_a[ty][tx] = 0.0f;
        }

        //load tile_b
        int b_row = tile * BLOCK_SIZE + ty;
        if (col < n && b_row < n) {
            tile_b[ty][tx] = b[b_row * n + col];
        }
        else {
            tile_a[ty][tx] = 0.0f;
        }

        __syncthreads(); // wait for tile_a and tile_b to be fully loaded

        for (int k = 0; k < BLOCK_SIZE; k++) {
            sum += tile_a[ty][k] * tile_b[k][tx];
        }
        __syncthreads(); // wait for sum to be calculated before moving tiles
    }

    if (row < n && col < n) { // in the case where more threads than elements
        c[row * n + col] = sum;
    }

}

int main() {
    int size = N * N * sizeof(float);
    float* A, float* B, float* C_cpu, float* C_gpu, float* C_gpu_tiled;
    float* d_A, float* d_B, float* d_C;
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C_cpu = (float*)malloc(size);
    C_gpu = (float*)malloc(size);
    C_gpu_tiled = (float*)malloc(size);

    for (int i = 0; i < N*N; i++) { //initializing
        A[i] = 2.0f;
        B[i] = 3.0f;
    }

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // cpu baseline
    auto start = std::chrono::high_resolution_clock::now();
    cpu_matrixmul(N, A, B, C_cpu);
    auto end = std::chrono::high_resolution_clock::now();
    auto microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
    std::cout << "CPU execution time for matrix mul: " << microseconds << " microseconds" << std::endl;

    // basic cuda
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((N + BLOCK_SIZE - 1)/BLOCK_SIZE, (N + BLOCK_SIZE - 1)/BLOCK_SIZE);

    start = std::chrono::high_resolution_clock::now();
    cuda_matrixmul<<<gridSize, blockSize>>>(N, d_A, d_B, d_C);
    end = std::chrono::high_resolution_clock::now();
    microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
    std::cout << "GPU execution time for matrix mul: " << microseconds << " microseconds" << std::endl;
    hipMemcpy(C_gpu, d_C, size, hipMemcpyDeviceToHost);

    // cuda tiled
    dim3 tiledblockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 tiledgridSize((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    start = std::chrono::high_resolution_clock::now();
    cuda_tiled_matrixmul<<<tiledblockSize, tiledgridSize>>>(N, d_A, d_B, d_C);
    end = std::chrono::high_resolution_clock::now();
    microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
    std::cout << "GPU execution time for tiled matrix mul: " << microseconds << " microseconds" << std::endl;
    hipMemcpy(C_gpu_tiled, d_C, size, hipMemcpyDeviceToHost);

    std::cout << C_cpu[0] << std::endl;
    std::cout << C_gpu[0] << std::endl;
    std::cout << C_gpu_tiled[0] << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C_cpu);
    free(C_gpu);
    free(C_gpu_tiled);
}